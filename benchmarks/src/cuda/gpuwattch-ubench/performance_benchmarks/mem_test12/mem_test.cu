#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <cutil.h>
#include <math.h>
// Includes
#include <stdio.h>
#include "../include/ContAcq-IntClk.h"
#include "../include/repeat2.h"
// includes, project
#include "../include/sdkHelper.h"  // helper for shared functions common to CUDA SDK samples
//#include <shrQATest.h>
//#include <shrUtils.h>

// includes CUDA
#include <hip/hip_runtime.h>

#define THREADS_PER_BLOCK 256
#define NUM_OF_BLOCKS 60

#define LINE_SIZE 	128
#define SETS		64
#define ASSOC		6
#define SIMD_WIDTH	32
#define NUM_OF_THREADS 32
// Variables
int* h_A;
int* h_B;
int* h_C;
int* d_A;
int* d_B;
int* d_C;
bool noprompt = false;
unsigned int my_timer;

// Functions
void CleanupResources(void);
void RandomInit(int*, int);
void ParseArguments(int, char**);

////////////////////////////////////////////////////////////////////////////////
// These are CUDA Helper functions

// This will output the proper CUDA error strings in the event that a CUDA host call returns an error
#define checkCudaErrors(err)  __checkCudaErrors (err, __FILE__, __LINE__)

inline void __checkCudaErrors(hipError_t err, const char *file, const int line ){
  if(hipSuccess != err){
	fprintf(stderr, "%s(%i) : CUDA Runtime API error %d: %s.\n",file, line, (int)err, hipGetErrorString( err ) );
	 exit(-1);
  }
}

// This will output the proper error string when calling hipGetLastError
#define getLastCudaError(msg)      __getLastCudaError (msg, __FILE__, __LINE__)

inline void __getLastCudaError(const char *errorMessage, const char *file, const int line ){
  hipError_t err = hipGetLastError();
  if (hipSuccess != err){
	fprintf(stderr, "%s(%i) : getLastCudaError() CUDA error : %s : (%d) %s.\n",file, line, errorMessage, (int)err, hipGetErrorString( err ) );
	exit(-1);
  }
}

// end of CUDA Helper Functions


int gcf(int a, int b)
{
        if (a == 0) return b;
        return gcf(b % a, a);
}


// Device code
const int page_size = 4;        // Scale stride and arrays by page size.

__global__ void global_latency (unsigned int * my_array, int array_length, int iterations, unsigned * duration) {

    
        unsigned  sum_time = 0;
        duration[0] = 0;
	unsigned j=0;
	unsigned LINESIZE= 1;
	unsigned CACHESIZE= 4096;
	unsigned LIMIT=0;
	int m=0;
/*
	// fill L1/L2 cache
	for (int k=0; k<CACHESIZE; k+=LINESIZE){
		m=k%array_length;
		j+=my_array[m];
	} 
	       
	if (j>=array_length) j=0;
*/
	int tid = blockDim.x * blockIdx.x + threadIdx.x;
	j=tid;
        for (int k = 0; k < iterations; k++) {
               repeat1(j = my_array[j];)
                // repeat1024(j=*(unsigned int **)j
        }

        //my_array[array_length] = (unsigned int)j;
        //my_array[array_length+1] = (unsigned int) sum_time;
        duration[0] = j;
}



void parametric_measure_global(int N, int iterations, int stride) {


        int i;
	int j=0;
        unsigned int * h_a;
        unsigned int * d_a;


        unsigned * duration;

        unsigned long long * latency;
        unsigned long long latency_sum = 0;

        // Don't die if too much memory was requested.
        if (N > 650000000) { printf ("OOM.\n"); return; }

        // allocate arrays on CPU 
        h_a = (unsigned int *)malloc(sizeof(unsigned int) * (N+2+NUM_OF_THREADS));
        latency = (unsigned long long *)malloc(sizeof(unsigned long long));

        // allocate arrays on GPU 
        hipMalloc ((void **) &d_a, sizeof(unsigned int) * (N+2+NUM_OF_THREADS));

        hipMalloc ((void **) &duration, sizeof(unsigned long long));

        // initialize array elements on CPU with pointers into d_a. 

        int step = gcf (stride, N);     // Optimization: Initialize fewer elements.
        for (i = 0; i < N; i += step) {
                // Device pointers are 32-bit on GT200.
                for (j=0; j<NUM_OF_THREADS; j++)
			h_a[i+j] = ((i + j + stride) % N);

        }
	for (j=0; j<NUM_OF_THREADS; j++)
		h_a[N+j] = j;
        h_a[N+NUM_OF_THREADS] = 0;
	

        hipDeviceSynchronize ();

        // copy array elements from CPU to GPU 
       hipMemcpy((void *)d_a, (void *)h_a, sizeof(unsigned int) * N, hipMemcpyHostToDevice);

        hipDeviceSynchronize ();


        // Launch a multiple of 10 iterations of the same kernel and take the average to eliminate interconnect (TPCs) effects 

        for (int l=0; l <1; l++) {

                // launch kernel
                dim3 Db = dim3(NUM_OF_THREADS);
                dim3 Dg = dim3(1,1,1);

                //printf("Launch kernel with parameters: %d, N: %d, stride: %d\n", iterations, N, stride); 
 
                global_latency <<<Dg, Db>>>(d_a,N, iterations, duration);

		//global_latency <<<Dg, Db>>> ();

                hipDeviceSynchronize ();

                hipError_t error_id = hipGetLastError();
                if (error_id != hipSuccess) {
                        printf("Error is %s\n", hipGetErrorString(error_id));
                }
		
                // copy results from GPU to CPU 
                hipDeviceSynchronize ();

                //hipMemcpy((void *)h_a, (void *)d_a, sizeof(unsigned int) * (N+2), hipMemcpyDeviceToHost);
                hipMemcpy((void *)latency, (void *)duration, sizeof(unsigned long long), hipMemcpyDeviceToHost);

                hipDeviceSynchronize ();
                latency_sum+=latency[0];

        }

        // free memory on GPU 
        hipFree(d_a);

        hipFree(duration);

        hipDeviceSynchronize ();

        // free memory on CPU 
        free(h_a);
        free(latency);


//	return 0;

}


	





// Host code
int main() {

 
	 printf("Assuming page size is %d KB\n", page_size);
        // we will measure latency of global memory
        // One thread that accesses an array.
        // loads are dependent on the previously loaded values

        int N, iterations, stride;

        // initialize upper bounds here
        int stride_upper_bound;

        printf("Global1: Global memory latency for 1 KB array and varying strides.\n");
        printf("   stride (bytes), latency (clocks)\n");


N= 536870912;
iterations = 40;
        stride_upper_bound = N;
stride= 2048;
	//for (stride = 1; stride <= (stride_upper_bound) ; stride+=1) {
        //        printf ("  %5d, ", stride*4);
          parametric_measure_global(N, iterations, stride);
        //}
    

        return 0;
}







